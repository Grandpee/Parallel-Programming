#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#define INF 10000000
#define V 10010

int vertexNum, edgeNum;
static int graphMap[V*V];
int *graphDist;
int B;

void input(char *inFileName);
void output(char *outFileName);

__global__ void cudaFW_phase1(int ith_round, int vertexNum, int *graph_dist, int B)
{
    extern __shared__ int dist[]; 

    int x = threadIdx.x;
    int y = threadIdx.y;
    int global_x = x + ith_round * B;//check if its x out of graphmap
    int global_y = y + ith_round * B;//check if its y out of graphmap

    //if k, q(coordinate in matrix graph_dist) smaller than vertexNum
    if (global_x < vertexNum && global_y < vertexNum)
        dist[x * B + y] = graph_dist[global_x * vertexNum + global_y];
    if (global_x >= vertexNum || global_y >= vertexNum)
        dist[x * B + y] = INF;
    __syncthreads();

    #pragma unroll
    for (int i=0; i<B; i++) {
        if (dist[x*B + y] > dist[x*B + i] + dist[i*B + y])
            dist[x*B + y] = dist[x*B + i] + dist[i*B + y];
        __syncthreads();
    }
    if (global_x < vertexNum && global_y < vertexNum)
        graph_dist[global_x * vertexNum + global_y] = dist[x * B + y];
    __syncthreads();
}

__global__ void cudaFW_phase2(int ith_round, int vertexNum, int *graph_dist, int B)
{
   if (blockIdx.x != ith_round) {// block not equal to pivot block
        extern __shared__ int share_space[];//two matrices
        int *first_matrix = &share_space[0];//pivot block
        int *second_matrix = &share_space[B * B];//self block

        int x = threadIdx.x;
        int y = threadIdx.y;
        int global_x = x + ith_round * B;//pivot x;
        int global_y = y + ith_round * B;//pivot y;

        //assign pivot block to first_matrix
        if (global_x < vertexNum && global_y < vertexNum)
            first_matrix[x*B + y] = graph_dist[global_x * vertexNum + global_y];
        if (global_x >= vertexNum || global_y >= vertexNum)
            first_matrix[x*B + y] = INF;
        

        if (blockIdx.y == 0)
            global_x = x + blockIdx.x * B;//row blocks, self coordinate
        if (blockIdx.y != 0)
            global_y = y + blockIdx.x * B;//column blocks, self coordinate

        if (global_x < vertexNum && global_y < vertexNum) {
            second_matrix[x*B + y] = graph_dist[global_x*vertexNum + global_y];
            __syncthreads();

            if (blockIdx.y == 0) {//row block
                #pragma unroll
                for (int i=0; i<B; i++) {
                    if (second_matrix[x*B + y] > second_matrix[x*B + i] + first_matrix[i*B + y])
                        second_matrix[x*B + y] = second_matrix[x*B + i] + first_matrix[i*B + y];
                    __syncthreads();
                }
            }
            if (blockIdx.y != 0) {//column block
                #pragma unroll
                for (int i=0; i<B; i++) {
                    if (second_matrix[x*B + y] > first_matrix[x*B + i] + second_matrix[i*B + y])
                        second_matrix[x*B + y] = first_matrix[x*B + i] + second_matrix[i*B + y];
                    __syncthreads();
                }
            }
            
            graph_dist[global_x * vertexNum + global_y] = second_matrix[x*B + y];
        }
    } 
}

__global__ void cudaFW_phase3(int ith_round, int vertexNum, int *graph_dist, int B, int blockOffset)
{
    int blockIdx_x = blockIdx.x + blockOffset;
    int blockIdx_y = blockIdx.y;
    if (blockIdx_x != ith_round && blockIdx_y != ith_round) {
        extern __shared__ int share_space[];
        int* row_block = &share_space[0];
        int* column_block = &share_space[B*B];

        int x = threadIdx.x;
        int y = threadIdx.y;
        int global_x = blockIdx_x * blockDim.x + x;
        int global_y = blockIdx_y * blockDim.y + y;
        int k = x + ith_round * B;//correspond row block possition
        int q = y + ith_round * B;//correspond column block position

        if (global_x < vertexNum && q < vertexNum)
            row_block[x*B + y] = graph_dist[global_x * vertexNum + q];
        if (global_x >= vertexNum || q >= vertexNum)
            row_block[x*B + y] = INF;
        if (global_y < vertexNum && k < vertexNum)
            column_block[x*B + y] = graph_dist[k * vertexNum + global_y];
        if (global_y >= vertexNum || k >= vertexNum)
            column_block[x*B + y] = INF;
        __syncthreads();

        if (global_x < vertexNum && global_y < vertexNum) {
            int selfDist = graph_dist[global_x * vertexNum + global_y];
            #pragma unroll
            for (int i=0; i<B; i++) {
                if (selfDist > row_block[x*B + i] + column_block[i*B + y])
                    selfDist = row_block[x*B + i] + column_block[i*B + y];
            }
            graph_dist[global_x * vertexNum + global_y] = selfDist;
        }
    }
}

__global__ void cuda_alignment (int *graph_dist, int *temp_dist, int blockOffset, int vertexNum)
{
    if (blockIdx.x >= blockOffset) {
        int x = blockIdx.x;
        for (int i=0; i<vertexNum; i++) {
            graph_dist[x * vertexNum + i] = temp_dist[x * vertexNum + i];
        }
    }

}

int main(int argc, char *argv[])
{
    if(argc != 4) {
        printf("not enough argument!\n");
        exit(0);
    }

    printf("I'm at starter\n");

    input(argv[1]);
    B = atoi(argv[3]);

    if (B > 32)
        B = 32;

    int round = ((vertexNum-1)+B) / B;
    graphDist = (int*) malloc (vertexNum * vertexNum * sizeof(int));


    int iDeviceCount = 0;
    hipGetDeviceCount( &iDeviceCount );
    if (iDeviceCount < 1) {
        printf("No GPU device\n");
        exit(0);
    }
    omp_set_num_threads(iDeviceCount);

    int *Dgraph_dist[iDeviceCount], *temp_dist[iDeviceCount];

    int blockOffset = ((round+1) + iDeviceCount) / iDeviceCount;
    
    dim3 blockNum_phase1(1,1);
    dim3 blockNum_phase2(round, 2);
    dim3 blockNum_phase3(blockOffset, round);
    dim3 threadNum(B, B);

    printf("I'm at here\n");


    #pragma omp parallel
    {
        unsigned int threadID = omp_get_thread_num();

        hipSetDevice(threadID);

	printf("I'm thead %d\n", threadID);

        hipMalloc((void**) &Dgraph_dist[threadID], vertexNum * vertexNum * sizeof(int));
        hipMalloc((void**) &temp_dist[threadID], vertexNum * vertexNum * sizeof(int));
        hipMemcpy(Dgraph_dist[threadID], graphMap, vertexNum * vertexNum * sizeof(int), hipMemcpyHostToDevice); 

        for (int i=0; i < round; i++) {

            #pragma omp barrier
            
            if (threadID == 0) {
                cudaFW_phase1 <<< blockNum_phase1, threadNum, B*B*sizeof(int) >>>(i, vertexNum, Dgraph_dist[threadID], B);
                cudaFW_phase2 <<< blockNum_phase2, threadNum, B*B*sizeof(int)*2 >>>(i, vertexNum, Dgraph_dist[threadID], B);
               
                hipMemcpy(Dgraph_dist[1], Dgraph_dist[0], vertexNum * vertexNum * sizeof(int), hipMemcpyDefault);
                hipStreamSynchronize(0);
	
            }

            #pragma omp barrier
            cudaFW_phase3 <<< blockNum_phase3, threadNum, B*B*sizeof(int)*2 >>>(i, vertexNum, Dgraph_dist[threadID], B, blockOffset * threadID);
            
            hipStreamSynchronize(0);

            if (threadID == 0) {
                hipMemcpy(temp_dist[0], Dgraph_dist[1], vertexNum * vertexNum * sizeof(int), hipMemcpyDefault);
                hipStreamSynchronize(0);
                cuda_alignment <<< vertexNum, 1 >>> (Dgraph_dist[0], temp_dist[0], B * blockOffset, vertexNum);
                hipStreamSynchronize(0);
            }
        }

    }
    hipMemcpy(graphDist, Dgraph_dist[0], vertexNum * vertexNum * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(Dgraph_dist[0]);
    hipFree(Dgraph_dist[1]);
    hipFree(temp_dist[0]);
    hipFree(temp_dist[1]);
    output(argv[2]);

    free(graphDist);

    exit(0);
}

void input(char *inFileName)
{
    FILE *infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &vertexNum, &edgeNum);

    for (int i=0; i<vertexNum; i++) {
        for (int j = 0; j < vertexNum; j++) {
            if (i == j)
                graphMap[i*vertexNum + j] = 0;
            else
                graphMap[i*vertexNum + j] = INF;
        }
    }

    while (--edgeNum >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        --a, --b;
        graphMap[a * vertexNum + b] = v;
    }
}

void output(char *outFileName)
{
    FILE *outfile = fopen(outFileName, "w");
    for (int i=0; i<vertexNum; i++) {
        for (int j=0; j<vertexNum; j++) {
            if (graphDist[i*vertexNum + j] >= INF)
                fprintf(outfile, "INF ");
            else
                fprintf(outfile, "%d ", graphDist[i*vertexNum + j]);
        }
        fprintf(outfile, "\n");
    }
}

